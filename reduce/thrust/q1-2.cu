#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>

using namespace std;

int main (int argc, char *argv[]) {
  // Basic error check
  if (argc != 2) {
    printf("Usage: %s N", argv[0]);
    exit(1);
  }

  // Get size
  int size = atoi(argv[1]);

  // Initialize vectors
  thrust::host_vector<float> h_data(size);
  thrust::host_vector<float> h_result(size);
  float h_sum, d_sum;

  // Timing varibles
  float time_reduce, time_scan;
  hipEvent_t start_reduce, end_reduce;
  hipEvent_t start_scan, end_scan;
  hipEventCreate(&start_reduce);
  hipEventCreate(&end_reduce);
  hipEventCreate(&start_scan);
  hipEventCreate(&end_scan);

  // Ready host and device data
  //thrust::generate(h_data.begin(), h_data.end(), rand);
  thrust::fill(h_data.begin(), h_data.end(), 1);

  // Reduction
  hipEventRecord(start_reduce, NULL);
  thrust::device_vector<float> d_data = h_data;

  d_sum = thrust::reduce(d_data.begin(), d_data.end(), (float)0, thrust::plus<float>());

  hipEventRecord(end_reduce, NULL);
  hipEventSynchronize(end_reduce);
  hipEventElapsedTime(&time_reduce, start_reduce, end_reduce);

  // Exclusive scan
  hipEventRecord(start_scan, NULL);
  thrust::device_vector<float> d_result = h_data;
  thrust::exclusive_scan(d_result.begin(), d_result.end(), d_result.begin());

  thrust::copy(d_result.begin(), d_result.end(), h_result.begin());

  hipEventRecord(end_scan, NULL);
  hipEventSynchronize(end_scan);
  hipEventElapsedTime(&time_scan, start_scan, end_scan);

  h_sum = thrust::reduce(h_data.begin(), h_data.end(), (float)0, thrust::plus<float>());
  thrust::exclusive_scan(h_data.begin(), h_data.end(), h_data.begin());
  
  cout << "Reduction time: " << time_reduce << endl;
  cout << "\tHost result: " << h_sum << endl;
  cout << "\tDevice result: " << d_sum << endl;

  if (thrust::equal(h_data.begin(), h_data.end(), h_result.begin())) {
  cout << "Prefix scan time: " << time_scan << endl;
  cout << "\tHost result: " << h_data[size-1] << endl;
  cout << "\tDevice result: " << h_result[size-1] << endl;
  }
  else {
    printf("Mismatch in scan results\n");
    // Only for debugging
    /*for(std::vector<float>::size_type i = 0; i != h_data.size(); i++) 
    {
      cout << h_data[i] << endl;
      cout << h_result[i] << endl;
    }*/
  }

  return 0;
}
