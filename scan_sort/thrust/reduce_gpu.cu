#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <time.h>
#include <random>

#define RUNS 20

using namespace std;

// Multiplier functor
struct multiplier{
  __host__ __device__
  double operator()(double x) {
    return (x * x);
  }
};

int main (int argc, char *argv[]) {
  // Basic error check
  if (argc != 2) {
    printf("Usage: %s N", argv[0]);
    exit(1);
  }

  // Select device
  hipSetDevice(1);

  // Get size
  int size = 1024 * 1024 * (10 + atoi(argv[1]));

  // Initialize vectors
  thrust::host_vector<double> h_data(size);
  thrust::host_vector<int> h_scan(size);
  thrust::host_vector<long> h_sort(size);
  //double h_sum, d_sum;

  // Timing varibles
  //float time_reduce;
  //hipEvent_t start_reduce, end_reduce;
  float time_sort, time_scan;
  hipEvent_t start_scan, end_scan;
  hipEvent_t start_sort, end_sort;
  //hipEvent_t start_scan, end_scan;
  //hipEventCreate(&start_reduce);
  //hipEventCreate(&end_reduce);
  hipEventCreate(&start_scan);
  hipEventCreate(&end_scan);
  hipEventCreate(&start_sort);
  hipEventCreate(&end_sort);

  // Ready host and device data
  srand(time(NULL));
  thrust::generate(h_data.begin(), h_data.end(), rand);
  //thrust::fill(h_data.begin(), h_data.end(), 1);

  // Reduction
  /*
  thrust::device_vector<double> d_data = h_data;
  d_sum = thrust::transform_reduce(d_data.begin(), d_data.end(), multiplier(), (double)0, thrust::plus<double>());
  
  hipEventRecord(start_reduce, NULL);
  
  for (int i=0; i<RUNS; i++) {
  d_sum = thrust::transform_reduce(d_data.begin(), d_data.end(), multiplier(), (double)0, thrust::plus<double>());
  }

  hipEventRecord(end_reduce, NULL);
  hipEventSynchronize(end_reduce);
  hipEventElapsedTime(&time_reduce, start_reduce, end_reduce);
  h_sum = thrust::transform_reduce(h_data.begin(), h_data.end(), multiplier(), (double)0, thrust::plus<double>());
  
  cout << "Reduction time: " << time_reduce/RUNS << " ms"<< endl;
  double time_sec = time_reduce / RUNS / 1e3;
  double gflops = 2 * size / time_sec / 1e9;
  cout << "N:" << size << "\tGFLOPS: " << gflops << endl;

  //cout << "\tHost result: " << h_sum << endl;
  //cout << "\tDevice result: " << d_sum << endl;
  double residue = (d_sum - h_sum) / h_sum;
  cout << "Residue: " << residue << endl;
  */


  // Exclusive scan
  std::default_random_engine rng( std::rand() );
  std::uniform_int_distribution<int> rnd_int;

  thrust::generate(h_scan.begin(), h_scan.end(), [&]() { return rnd_int(rng); });
  thrust::device_vector<int> d_scan = h_scan;
  thrust::exclusive_scan(d_scan.begin(), d_scan.end(), d_scan.begin());
  
  hipEventRecord(start_scan, NULL);

  for (int i=0; i<RUNS; i++)
      thrust::exclusive_scan(d_scan.begin(), d_scan.end(), d_scan.begin());

  hipEventRecord(end_scan, NULL);
  hipEventSynchronize(end_scan);
  hipEventElapsedTime(&time_scan, start_scan, end_scan);

  //thrust::copy(d_scan.begin(), d_scan.end(), h_scan.begin());
  //thrust::exclusive_scan(h_data.begin(), h_data.end(), h_data.begin());
  
  cout << "Scan time: " << time_scan/RUNS << " ms"<< endl;
  double time_sec = time_scan / RUNS / 1e3;
  cout << "Scan N:" << size << "\tkeys/sec: " << size / time_sec << endl;

  /*
  if (thrust::equal(h_data.begin(), h_data.end(), h_scan.begin())) {
  cout << "Prefix scan time: " << time_scan << endl;
  cout << "\tHost result: " << h_data[size-1] << endl;
  cout << "\tDevice result: " << h_result[size-1] << endl;
  }
  else {
    printf("Mismatch in scan results\n");
    // Only for debugging
    for(std::vector<double>::size_type i = 0; i != h_data.size(); i++) 
    {
      cout << h_data[i] << endl;
      cout << h_result[i] << endl;
    }
  }
  */

  // Sort
  std::uniform_int_distribution<long> rnd_long;

  thrust::generate(h_sort.begin(), h_sort.end(), [&]() { return rnd_long(rng); });
  thrust::device_vector<long> d_sort = h_sort;
  thrust::sort(d_sort.begin(), d_sort.end());
  
  hipEventRecord(start_sort, NULL);

  for (int i=0; i<RUNS; i++)
      thrust::sort(d_sort.begin(), d_sort.end());

  hipEventRecord(end_sort, NULL);
  hipEventSynchronize(end_sort);
  hipEventElapsedTime(&time_sort, start_sort, end_sort);

  cout << "Sort time: " << time_sort/RUNS << " ms"<< endl;
  time_sec = time_sort / RUNS / 1e3;
  cout << "Sort N:" << size << "\tkeys/sec: " << size / time_sec << endl;

  return 0;
}
