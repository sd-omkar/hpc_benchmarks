#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <iostream>
#include <iterator>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdio>
#include <time.h>

#define A 5.f
#define RUNS 25

using namespace std;

struct saxpy_functor: public thrust::binary_function<float,float,float> {
    const float a;
    saxpy_functor(float _a) : a(_a) {}
    __host__ __device__
    float operator()(const float &x, const float &y) const { 
        return a * x + y;
    }
};

int main (int argc, char *argv[]) {
    // Check argument
    if (argc != 2) {
        printf("Usage: %s N", argv[0]);
        exit(1);
    }

    // Select device
    hipSetDevice(0);

    // Get size
    int size = 1024 * 1024 * (10 + atoi(argv[1]));

    // Initialize host vectors
    thrust::host_vector<float> h_X(size);
    thrust::host_vector<float> h_Y(size);

    // Timing variables
    float time_saxpy;
    hipEvent_t start_saxpy, end_saxpy;
    hipEventCreate(&start_saxpy);
    hipEventCreate(&end_saxpy);

    // Generate vectors
    srand(time(NULL));
    thrust::generate(h_X.begin(), h_X.end(), rand);
    thrust::generate(h_Y.begin(), h_Y.end(), rand);

    // SAXPY
    // Y <- A*X + Y 
    thrust::device_vector<float> d_X = h_X;
    thrust::device_vector<float> d_Y = h_Y;
    thrust::transform(d_X.begin(), d_X.end(),
                      d_Y.begin(), d_Y.begin(),
                      saxpy_functor(A));

    hipEventRecord(start_saxpy, NULL);

    for (int i=0; i<RUNS; i++)
        thrust::transform(d_X.begin(), d_X.end(),
                          d_Y.begin(), d_Y.begin(),
                          saxpy_functor(A));

    hipEventRecord(end_saxpy, NULL);
    hipEventSynchronize(end_saxpy);
    hipEventElapsedTime(&time_saxpy, start_saxpy, end_saxpy);

    cout << "SAXPY time: " << time_saxpy / RUNS << " ms" << endl;
    double time_sec = time_saxpy / RUNS / 1e3;
    double gflops = 2 * size / time_sec / 1e9;
    cout << "N: " << size << "\tGFLOPS: " << gflops << endl;

    return 0;
}
