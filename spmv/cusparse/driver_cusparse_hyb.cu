#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <cmath>
#include <map>
#include <stdio.h>
#include <stdlib.h>

#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>

#include <timer.h>

#include "hipsparse.h"


// -----------------------------------------------------------------------------
// Macro to obtain a random number between two specified values
// -----------------------------------------------------------------------------
#define RAND(L,H)  ((L) + ((H)-(L)) * (float)rand()/(float)RAND_MAX)


// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------
#ifdef WIN32
#   define ISNAN(A)  (_isnan(A))
#else
#   define ISNAN(A)  (isnan(A))
#endif


// -----------------------------------------------------------------------------
// Typedefs
// -----------------------------------------------------------------------------
typedef double REAL;
typedef double PREC_REAL;

typedef typename cusp::csr_matrix<int, REAL, cusp::device_memory> Matrix;
typedef typename cusp::array1d<REAL, cusp::device_memory>         Vector;
typedef typename cusp::array1d<REAL, cusp::host_memory>           VectorH;
typedef typename cusp::array1d<PREC_REAL, cusp::device_memory>    PrecVector;


// -----------------------------------------------------------------------------
using std::cout;
using std::cerr;
using std::cin;
using std::endl;
using std::string;
using std::vector;

void spmv(hipsparseHybMat_t& hybA,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const float *x, float *y);

void spmv(hipsparseHybMat_t& hybA,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const double *x, double *y);

// -----------------------------------------------------------------------------
// MAIN
// -----------------------------------------------------------------------------
int main(int argc, char** argv) 
{
	// Set up the problem to be solved.
	string         fileMat;
	if (argc < 2) {
		cerr << "Usage: ./driver_cusparse_hyb ${MATRIX_FILE_NAME}" << endl;
		exit(-1);
	}

	fileMat = argv[1];

	//cout << fileMat << endl;

	// Get matrix and rhs.
	Matrix A;
	Vector b;
	Vector x;

	cusp::io::read_matrix_market_file(A, fileMat);

	b.resize(A.num_rows);

	{
		VectorH x_h(A.num_rows);

		for (int i = 0; i < A.num_rows; i++)
			x_h[i] = RAND(2,10) / 2;

		x = x_h;
	}

	hipsparseHandle_t handle;
	hipsparseCreate(&handle);

	hipsparseHybMat_t hybA;
	hipsparseCreateHybMat(&hybA);

	hipsparseMatDescr_t descrA;
	hipsparseCreateMatDescr(&descrA);

	hipsparseSetMatType(descrA,      HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatDiagType(descrA,  HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrA,  HIPSPARSE_FILL_MODE_LOWER);

	hipsparseDcsr2hyb(handle, A.num_rows, A.num_rows, descrA, thrust::raw_pointer_cast(&A.values[0]), thrust::raw_pointer_cast(&A.row_offsets[0]), thrust::raw_pointer_cast(&A.column_indices[0]), hybA, A.num_entries, HIPSPARSE_HYB_PARTITION_AUTO);

	CUDATimer timer;
	int counter = 0;
	double elapsed = 0.0;
	for (int i = 0; i < 10; i++) {
		timer.Start();
		spmv(hybA, handle, descrA, thrust::raw_pointer_cast(&x[0]), thrust::raw_pointer_cast(&b[0]));
		timer.Stop();

		if (i > 0) {
			counter ++;
			elapsed += timer.getElapsed();
		}
	}
	elapsed /= counter;
	//cout << "cuSparse HYB: " << elapsed << endl;

	long int rows,cols,temp,NNZ=0;
	std::ifstream infile(fileMat.c_str());
	string line;
	bool flag=false;
	while (std::getline(infile, line)){
		if(!flag && line.substr(0,1).compare("%")!=0){
			std::istringstream iss(line);
			iss>>rows>>cols>>temp;
    			flag=true;
		}	
		if(flag)
			NNZ++;
	}

	cout<<"name = SPMV"<<endl;
	while(fileMat.find("/")!=string::npos){
		fileMat = fileMat.substr(fileMat.find("/")+1,fileMat.length());
	}
        cout<<"input = "<<fileMat<<endl;
	cout<<"datatype = double"<<endl;
  	cout<<"dim_x = "<<rows<<endl;
	cout<<"dim_y = "<<cols<<endl;
	cout<<"NNZ = "<<NNZ-1<<endl;
	cout<<"value_type = GFLOPS"<<endl;
	cout<<"value = "<<(2*NNZ*1e-9)/(elapsed*1e-3)<<endl;

	hipsparseDestroyMatDescr(descrA);
	hipsparseDestroyHybMat(hybA);
	hipsparseDestroy(handle);

	return 0;
}

void spmv(hipsparseHybMat_t& hybA,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const float *x, float *y)
{
	float one = 1.f, zero = 0.f;
	hipsparseShybmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, descrA, hybA, x, &zero, y);
}

void spmv(hipsparseHybMat_t& hybA,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const double *x, double *y)
{
	double one = 1.0, zero = 0.0;
	hipsparseDhybmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, descrA, hybA, x, &zero, y);
}
