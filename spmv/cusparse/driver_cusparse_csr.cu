#include "hip/hip_runtime.h"
#include <algorithm>
#include <fstream>
#include <cmath>
#include <map>
#include <stdio.h>
#include <stdlib.h>

#include <cusp/io/matrix_market.h>
#include <cusp/csr_matrix.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>

#include <timer.h>

#include "hipsparse.h"


// -----------------------------------------------------------------------------
// Macro to obtain a random number between two specified values
// -----------------------------------------------------------------------------
#define RAND(L,H)  ((L) + ((H)-(L)) * (float)rand()/(float)RAND_MAX)


// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------
#ifdef WIN32
#   define ISNAN(A)  (_isnan(A))
#else
#   define ISNAN(A)  (isnan(A))
#endif


// -----------------------------------------------------------------------------
// Typedefs
// -----------------------------------------------------------------------------
typedef double REAL;
typedef double PREC_REAL;

typedef typename cusp::csr_matrix<int, REAL, cusp::device_memory> Matrix;
typedef typename cusp::array1d<REAL, cusp::device_memory>         Vector;
typedef typename cusp::array1d<REAL, cusp::host_memory>           VectorH;
typedef typename cusp::array1d<PREC_REAL, cusp::device_memory>    PrecVector;


// -----------------------------------------------------------------------------
using std::cout;
using std::cerr;
using std::cin;
using std::endl;
using std::string;
using std::vector;

void spmv(int n, int nnz, const int *row_offsets, const int *column_indices, const float *values,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const float *x, float *y);

void spmv(int n, int nnz, const int *row_offsets, const int *column_indices, const double *values,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const double *x, double *y);

// -----------------------------------------------------------------------------
// MAIN
// -----------------------------------------------------------------------------
int main(int argc, char** argv) 
{
	// Set up the problem to be solved.
	string         fileMat;
	if (argc < 2) {
		cerr << "Usage: ./driver_cusparse_csr ${MATRIX_FILE_NAME}" << endl;
		exit(-1);
	}

	fileMat = argv[1];

	cout << fileMat << endl;

	// Get matrix and rhs.
	Matrix A;
	Vector b;
	Vector x;

	cusp::io::read_matrix_market_file(A, fileMat);

	b.resize(A.num_rows);

	{
		VectorH x_h(A.num_rows);

		for (int i = 0; i < A.num_rows; i++)
			x_h[i] = RAND(2,10) / 2;

		x = x_h;
	}

	hipsparseHandle_t        handle;
	hipsparseMatDescr_t      descrA;

	hipsparseCreate(&handle);
	hipsparseCreateMatDescr(&descrA);

	hipsparseSetMatType(descrA,      HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatDiagType(descrA,  HIPSPARSE_DIAG_TYPE_NON_UNIT);
	hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrA,  HIPSPARSE_FILL_MODE_LOWER);

	CUDATimer timer;
	int counter = 0;
	double elapsed = 0.0;
	for (int i = 0; i < 10; i++) {
		timer.Start();
		spmv(A.num_rows, A.num_entries, thrust::raw_pointer_cast(&A.row_offsets[0]), thrust::raw_pointer_cast(&A.column_indices[0]), 
			 thrust::raw_pointer_cast(&A.values[0]), handle, descrA, thrust::raw_pointer_cast(&x[0]), thrust::raw_pointer_cast(&b[0]));
		timer.Stop();

		if (i > 0) {
			counter ++;
			elapsed += timer.getElapsed();
		}
	}
	elapsed /= counter;
	cout << "cuSparse CSR: " << elapsed << endl;

	hipsparseDestroyMatDescr(descrA);
	hipsparseDestroy(handle);

	return 0;
}

void spmv(int n, int nnz, const int *row_offsets, const int *column_indices, const float *values,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const float *x, float *y)
{
	float one = 1.f, zero = 0.f;
    hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, values, row_offsets, column_indices, x, &zero, y);
}

void spmv(int n, int nnz, const int *row_offsets, const int *column_indices, const double *values,
		  hipsparseHandle_t& handle, hipsparseMatDescr_t& descrA,
		  const double *x, double *y)
{
	double one = 1.0, zero = 0.0;
    hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nnz, &one, descrA, values, row_offsets, column_indices, x, &zero, y);
}
